#include <vector>

#include "caffe/layers/base_data_layer.hpp"

namespace caffe {

template <typename Dtype>
void BasePrefetchingDataLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  Batch<Dtype>* batch = prefetch_full_.pop("Data layer prefetch queue empty");
  // Reshape to loaded data.
  top[0]->ReshapeLike(batch->data_);
  // Copy the data
  caffe_copy(batch->data_.count(), batch->data_.gpu_data(),
      top[0]->mutable_gpu_data());
  if (this->output_labels_) {
    // Reshape to loaded labels.
    top[1]->ReshapeLike(batch->label_);
    // Copy the labels.
    caffe_copy(batch->label_.count(), batch->label_.gpu_data(),
        top[1]->mutable_gpu_data());
  }
  // Ensure the copy is synchronous wrt the host, so that the next batch isn't
  // copied in meanwhile.
  CUDA_CHECK(hipStreamSynchronize(hipStreamDefault));
  prefetch_free_.push(batch);
}

template <typename Dtype>
void BaseROIPrefetchingDataLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {

    BatchROI<Dtype>* batch = prefetch_roi_full_.pop("Data layer prefetch queue empty");
    //Reshape to loaded data
    top[0]->ReshapeLike(batch->data_);
    caffe_copy(batch->data_.count(), batch->data_.gpu_data(), top[0]->mutable_gpu_data());
    if(this->output_labels_)
    {
        top[2]->ReshapeLike(batch->label_);
        caffe_copy(batch->label_.count(), batch->label_.gpu_data(), top[2]->mutable_gpu_data());
    }
    top[1]->ReshapeLike(batch->rois_);
    caffe_copy(batch->rois_.count(), batch->rois_.gpu_data(), top[1]->mutable_gpu_data());
    
    top[3]->ReshapeLike(batch->bboxes_target_);
    caffe_copy(batch->bboxes_target_.count(), batch->bboxes_target_.gpu_data(), top[3]->mutable_gpu_data());
    
    top[4]->ReshapeLike(batch->bboxes_weight_);
    caffe_copy(batch->bboxes_weight_.count(), batch->bboxes_weight_.gpu_data(), top[4]->mutable_gpu_data());
    
    // Ensure the copy is synchronous wrt the host, so that the next batch isn't
    // copied in meanwhile.
    CUDA_CHECK(hipStreamSynchronize(hipStreamDefault));
    prefetch_roi_free_.push(batch);
}

INSTANTIATE_LAYER_GPU_FORWARD(BasePrefetchingDataLayer);
INSTANTIATE_LAYER_GPU_FORWARD(BaseROIPrefetchingDataLayer);

}  // namespace caffe
